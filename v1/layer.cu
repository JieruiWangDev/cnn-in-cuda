#include "hip/hip_runtime.h"
#include "layer.h"


#define TILE_WIDTH 16


// Layer constructor:
Layer::Layer(int in_width, int in_height, int in_size): M(in_width), N(in_height), bytes(in_size){

    float h_bias[N];
    float h_weight[N][M];


    output = NULL;
    preact = NULL;
    bias = NULL;
    weight = NULL;

    for (int i = 0; i < N; i++){
        h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);  // initial bias
        for (int j = 0; j < M; j++){
            h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);  // initial weight
        }
    }

    hipMalloc(&output, sizeof(float) * bytes);
	hipMalloc(&preact, sizeof(float) * bytes);

	hipMalloc(&bias, sizeof(float) * N);
	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * bytes);
	hipMalloc(&d_preact, sizeof(float) * bytes);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// de-constructor
Layer::~Layer(){

    // TODO: free cuda memory
    hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);

}

void Layer:: setInput(float *data){
    hipMemcpy(output, data, sizeof(float)*bytes, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * bytes);
	hipMemset(preact, 0x00, sizeof(float) * bytes);
}


void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * bytes);
	hipMemset(d_preact, 0x00, sizeof(float) * bytes);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}


__device__ float sigmoid(float s){
    return 1/(1 + exp(-s));
}

__global__ void apply_sigmoid(float *input, float *output, const int N){
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    int size = blockDim.x * gridDim.x;
    // TODO:
    for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = sigmoid(input[idx]);
	}
}

// __global__ void backward_sigmoid(float* X, int size_in)
// {
// 	int t = blockIdx.x * 1024 + threadIdx.x;

// 	if(t < size_in)
// 	{
// 		double tmp = 1 / (1 + exp(-X[t]));
// 		tmp = (1-tmp)*tmp;
// 		X[t] = X[t]*tmp;
// 	}
// }


__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
	}
}

#define TILE_WIDTH 16

//input_pointer,  Output_pointer, W_pointer, Inputimage_channel, Inputimage_height, Inputimage_width , Outputimage_width, W_width_height, Outputimage_channel
__global__ void ConvLayerForward_Kernel_1(float input[28][28], float output[6][24][24], float weight[6][5][5], float bias[6], int C, int H_in, int W_in, int W_out, int K, int M){
    int H_out = H_in - K + 1;
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0)
		W_grid = 1;
	// int l = blockIdx.x;
	int m = blockIdx.y;
	int x = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
	int y = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

	float acc = 0;
	int c, p, q;
	for (c = 0; c < C; c++) { // sum over all input channels
		for (p = 0; p < K; p++) // loop over KxK filter
			for (q = 0; q < K; q++)
				if(x < H_out && y < W_out)
                    acc += input[x+p][y+q] * weight[m][p][q];
					//acc = acc + X[n*(C*H_in*W_in) + c*(H_in*W_in) + (hx+p)*(W_in) + (y+q)] * W[m*(C*K*K) + c*(K*K) + p*(K) + q];
	}
	__syncthreads();
	if(x < H_out && y < W_out)
	{
        output[m][x][y] = acc + bias[m];
    }
}


// input_pointer, output_pointer, inputimage_height, inputimage_width, outputimage_channel, pool_size 
__global__ void MaxPool2dForward_Kernel_1(float input[6][24][24], float output[6][6][6], float weight[1][4][4], float bias[1] ,int H_in, int W_in, int M, int pool_size){
	int H_out = H_in/pool_size;
	int W_out = W_in/pool_size;
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0){
        W_grid = 1;
    }
		
	// int l = blockIdx.x;
	int m = blockIdx.y;
	int x = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
	int y = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;
	//h and w is not center point of calculating, it's upper left corner point of Input image
	
	float acc = 0;
	int p, q;
	for (p = 0; p < pool_size; p++) { // loop over KxK input samples
		for (q = 0; q < pool_size; q++)
			if(x < H_out && y < W_out)
				// acc = acc + input[l*(M*H_in*W_in)+ m*(H_in*W_in) +
				//               (pool_size * x + p)*(W_in) + (pool_size * y + q)] / (pool_size * pool_size);
                acc = acc + input[m][pool_size * x+p][pool_size * y+q] * weight[0][p][q];
	}
	__syncthreads();
	if(x < H_out && y < W_out)
	{
		// Y[n*(M*H_out*W_out)+ m*(H_out*W_out) + h*(W_out) + w] = acc;
		output[m][x][y] = acc + bias[0];
	}
}


__global__ void FullyConLayerForward_kernel(float input[6][6][6], float weight[10][6][6][6], float output[10], float bias[10], int H_in, int W_in, int W_we , int H_out, int W_out) {
	int W_grid = ceilf((float)W_out/TILE_WIDTH);
	if(W_grid==0)
		W_grid = 1;

	// int n = blockIdx.x;
	int m = blockIdx.y;  // 10
	int h = threadIdx.x;  // 6
	int w = threadIdx.y;  // 6
	int y = threadIdx.z;  // 6

	float Pvalue = 0;
	int o, p, q;
	for (o = 0; o < 6; o++) {
		for (p = 0; p < 6; p++) {
			for (q = 0; q < 6; q++){
				if(h < 6 && w < 6 && y < 6)
				// Pvalue += input[y][h+p][w+q] * weight[m][y][h+p][w+q];
				// Pvalue += input[h][w][y] * weight[m][h+o][w+p][y+q];
				Pvalue+= input[o][p][q] * weight[m][o][p][q];
			}
		}
	}
	__syncthreads();

    if(m < W_out && h < 6 && w < 6 && y < 6)
		output[m] = Pvalue + bias[m]; // Output
}

// input_height, input_width, weight_width, output_height, output_width
//      1             6          10          1              10
// __global__ void FullyConLayerForward_kernel(float input[6][6][6], float weight[10][6][6][6], float output[10], float bias[10], int H_in, int W_in, int W_we , int H_out, int W_out) {
// 	int W_grid = ceilf((float)W_out/TILE_WIDTH);
// 	if(W_grid==0)
// 		W_grid = 1;

// 	// int l = blockIdx.x;
// 	int m = blockIdx.y;  // 10
// 	// int x = threadIdx.x;
// 	// int y = threadIdx.y;
// 	// int z = threadIdx.z;

// 	float Pvalue = 0;
// 	int o, p, q;
// 	for (o = 0; o < 6; o++) {
// 		for (p = 0; p < 6; p++) {
// 			for (q = 0; q < 6; q++){
// 				Pvalue += input[o][p][q] * weight[m][o][p][q];
// 			}
// 		}
// 	}
// 	__syncthreads();

// 	if(m < W_out)
// 		output[m] = Pvalue + bias[m]; // Output

// 	// float Pvalue = 0;
// 	// for (int i = 0; i < 6; i++){
// 	// 	if(x < 6 && y < 6)
// 	// 	Pvalue += input[x][y][i] * weight[m][x][y][i];
// 	// }
// 	// // __syncthreads();

//     // if(m < 10 && x < 6 && y < 6)
// 	// 	atomicAdd(&output[m], Pvalue);
// 	// if(x==0 && y==0)
// 	// 	output[m] += bias[m];
// }


// __global__ void FullyConLayerBackward_kernel(
// 	float lf_output[10],
// 	float l_f_d_preact[10],
// 	float ls1_preact[6][6][6],
// 	float lf_weight[10][6][6][6],
// 	float lf_d_weight[10][6][6][6],
// 	float lf_bias[10]
// ) {
// 	// int l = blockIdx.x;
// 	int m = blockIdx.y;  // 10
// 	int x = threadIdx.x;  // 6
// 	int y = threadIdx.y;  // 6
// 	int z = threadIdx.z;  // 6

// 	l_f_d_preact[m] *= lf_output[m] * (1- lf_output[m]);
// 	__syncthreads();
// 	// ls1_d_preact[m] = l_f_d_preact[m] * lf_output[m] * (1- lf_output[m]);

// 	lf_bias[m] += lr + l_f_d_preact[m];
	
// 	lf_d_weight[m][x][y][z] = l_f_d_preact[m] * ls1_preact[x][y][z] ;
// 	lf_d_weight[m][x][y][z] += lf_weight[m][x][y][z];
// }


// //input_pointer, Inputimage_height, Inputimage_width, output_pointer, Outputimage_channel, pool_size
// __global__ void poolingLayer_backward_GPU(float input[6][24][24], int H_in, int W_in, float output[6][6][6], int M, int pool_size)

// {
// 	int H_out = H_in/pool_size;
// 	int W_out = W_in/pool_size;
// 	int W_grid = ceilf((float)W_out/TILE_WIDTH);
// 	if(W_grid==0)
// 		W_grid = 1;
// 	// int l = blockIdx.x;
// 	int m = blockIdx.y;
// 	int x = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
// 	int y = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

// 	//h and w is not center point of calculating, it's upper left corner point of Input image
// 	float acc = 0;
// 	for (int p = 0; p < pool_size; p++) { // loop over KxK input samples
// 		for (int q = 0; q < pool_size; q++)
// 			if(x < H_out && y < W_out)
// 			input[m][h+p][w+q] = output[m][x][y] / (pool_size * pool_size);
// 	}
// 	__syncthreads();

// }



// __global__ void ConvLayerBackward_Kernel(
// 	float input[28][28], 
// 	float d_output[6][24][24], 
// 	float preact[6][24][24], 
// 	float d_preact[6][24][24], 
// 	float d_weight[6][5][5], 
// 	int C, int H_in, int W_in, int W_out, int K, int M) {

//     int H_out = H_in - K + 1;
// 	int c, p, q;
// 	int W_grid = ceilf((float)W_out/TILE_WIDTH);
// 	if(W_grid==0)
// 		W_grid = 1;
// 	int l = blockIdx.x;
// 	int m = blockIdx.y;
// 	int x = (blockIdx.z / W_grid)*TILE_WIDTH + threadIdx.y;
// 	int y = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

// 	float d = 24.0f * 24.0f;

// 	float o = sigmoid(preact[m][x][y]);
	
// 	// float dv = d_output[m][x][y] * o * (1 - o);
// 	d_preact[m][x][y] = d_output[m][x][y] * o * (1 - o);
// 	__syncthreads();

// 	for (c = 0; c < C; c++) {
// 		for (p = 0; p < K; p++) {
// 			for (q = 0; q < K; q++) {
// 				if(x < H_out && y < W_out) {
// 					d_weight[m][p][q] = d_preact[m][x][y] * input[28][28]/d;
// 				}
// 			}
// 		}
// 	}
// }


__global__ void bp_f(
	float l_f_d_weight[10][6][6][6],
	float l_f_d_preact[10],
	float l_f_bias[10],
	float l_f_weight[10][6][6][6],
	float l_s1_output[6][6][6],
	float l_s1_d_output[6][6][6],
	float l_s1_d_preact[6][6][6]
){
	// int l = blockIdx.x;
	int m = blockIdx.y;  // 10
	int x = threadIdx.x;  // 6
	int y = threadIdx.y;  // 6
	int z = threadIdx.z;  // 6


	l_f_d_weight[m][x][y][z] = l_f_d_preact[m] * l_s1_output[x][y][z];
	// l_s1_d_output[x][y][z] += l_f_weight[m][x][y][z] * l_f_d_preact[m];

	atomicAdd(&l_s1_d_output[x][y][z], l_f_weight[m][x][y][z] * l_f_d_preact[m]);
	if(x==0 && y==0 && z==0 )
		l_f_bias[m] += lr * l_f_d_preact[m];

	l_f_weight[m][x][y][z] += lr * l_f_d_weight[m][x][y][z];
}

__global__ void bp_s1(
	float l_s1_preact[6][6][6],
	float l_s1_d_output[6][6][6],
	float l_s1_d_preact[6][6][6],
	float l_s1_d_weight[1][4][4],
	float l_s1_weight[1][4][4],
	float l_c1_output[6][24][24],
	float l_c1_d_output[6][24][24],
	float l_s1_bias[6]
){
	// int l = blockIdx.x;
	int m = blockIdx.y;  // 6
	int x = threadIdx.x;  // 6
	int y = threadIdx.y;  // 6
	// int z = threadIdx.z;

	float o = sigmoid(l_s1_preact[m][x][y]);
	l_s1_d_preact[m][x][y] = l_s1_d_output[m][x][y] * o * (1 - o);

	// l_s1_d_preact[m][x][y] = l_s1_d_output[m][x][y] * l_s1_output[m][x][y] * (1 - l_s1_output[m][x][y]);
	__syncthreads();

	l_s1_bias[0] += lr * l_s1_d_preact[m][x][y]/(6*6*6);

	int i,j;
	for(i=0; i<4; i++) {
		for(j=0; j<4; j++) {
			// l_s1_d_weight[0][i][j] += l_s1_d_preact[m][x][y] * l_c1_output[m][h*4+i][w*4+j];
			// l_c1_d_output[m][h*4+i][w*4+j] += l_s1_weight[0][i][j] * l_s1_d_preact[m][x][y];

			atomicAdd(&l_s1_d_weight[0][i][j], l_s1_d_preact[m][x][y] * l_c1_output[m][x*4+i][y*4+j]);
			atomicAdd(&l_c1_d_output[m][x*4+i][y*4+j], l_s1_weight[0][i][j] * l_s1_d_preact[m][x][y]);
		}
	}

	if(m==0 && x<4 && y<4)
		l_s1_weight[0][x][y] += lr * l_s1_d_weight[0][x][y];
}


__global__ void bp_c1(
	float l_c1_preact[6][24][24],
	float l_c1_d_preact[6][24][24],
	float l_c1_d_output[6][24][24],
	float l_c1_d_weight[6][5][5],
	float l_c1_weight[6][5][5],
	float l_input_output[28][28],
	float l_c1_bias[6]
){
	// int l = blockIdx.x;
	int m = blockIdx.y;  // 6
	int x = threadIdx.x;  // 24
	int y = threadIdx.y;  // 24
	// int z = threadIdx.z;


	float o = sigmoid(l_c1_preact[m][x][y]);
	l_c1_d_preact[m][x][y] = l_c1_d_output[m][x][y] * o * (1 - o);

	int i, j;
	for(i=0; i<5; i++){
		for(j=0; j<5; j++){
			l_c1_d_weight[m][i][j] += l_c1_d_preact[m][x][y] * l_input_output[x + i][y + j] / (24*24);
		}
	}

	l_c1_bias[m] += lr * l_c1_d_preact[m][x][y] / (6*24*24);

	if(m==6 && x<5 && y<5)
		l_c1_weight[m][x][y] += lr * l_c1_d_weight[m][x][y];
}
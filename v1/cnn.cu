#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define USE_MNIST_LOADER
#define MNIST_DOUBLE

// includes, system
#include <string>

#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <math.h>
#include <time.h>

#include "layer.h"
#include "layer.cu"

struct mnist_data {
	double data[28][28];
	int label;  //0-9
};

//define the kernel size
#define TILE_WIDTH 16  //for small example

// set Layer
static Layer l_input = Layer(0, 0, 28*28);
static Layer l_c1 = Layer(5*5, 6, 24*24*6);
static Layer l_s1 = Layer(4*4, 1, 6*6*6);
static Layer l_f = Layer(6*6*6, 10, 10);

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

unsigned int dataToInt(char* c) {
	unsigned int d = 0;
	for (int i = 0; i < 4; i++) {
		d <<= 8;
		d |= (unsigned char)c[i];
	}
	return d;
}

int mnist_load(
    const char *image_filename,
	const char *label_filename,
	mnist_data **data,
	unsigned int *count) 
{
    char tmp[4];
    unsigned char read_data[28*28];
    unsigned int im, l, i, j, k, ic1, ic2, image_cnt, label_cnt;

    FILE *ifp = fopen(image_filename, "rb");
	FILE *lfp = fopen(label_filename, "rb");

    if (!ifp || !lfp) {
        printf("file not open");
        if (ifp) fclose(ifp);
        if (lfp) fclose(lfp);
        return -1;
    }

    fread(tmp, 1, 4, ifp);
	im = dataToInt(tmp);
	fread(tmp, 1, 4, lfp);
	l = dataToInt(tmp);
    fread(tmp, 1, 4, ifp);
	image_cnt = dataToInt(tmp);
	fread(tmp, 1, 4, lfp);
	label_cnt = dataToInt(tmp);

    fread(tmp, 1, 4, ifp);
	ic1 = dataToInt(tmp);
    fread(tmp, 1, 4, ifp);
	ic2 = dataToInt(tmp);

    // printf("im, l, image_cnt, label_cnt, ic1, ic2 \n");
    // printf("%d, %d, %d, %d, %d, %d \n", im, l, image_cnt, label_cnt, ic1, ic2);

    if(im != 2051 || l != 2049 || image_cnt != label_cnt || ic1 != 28 || ic2 != 28){
        printf("get wrong file");
        fclose(ifp);
        fclose(lfp);
        return -2;
    }

    *count = image_cnt;
	*data = (mnist_data *)malloc(sizeof(mnist_data) * image_cnt);

    for (i = 0; i < image_cnt; i++) {
        mnist_data *d = &(*data)[i];

        fread(read_data, 1, 28*28, ifp);
        for(j=0; j<28; j++){
            for(k=0; k<28; k++)
                d->data[j][k] = read_data[j*28+k]/255.0;
        }

        fread(tmp, 1, 1, lfp);
		d->label = tmp[0]%10;
    }
    fclose(ifp);
    fclose(lfp);
    return 0;
}

static inline void loadData(){
    mnist_load("MNIST_data/train-images.idx3-ubyte", "MNIST_data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("MNIST_data/t10k-images.idx3-ubyte", "MNIST_data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);
}

static float forward(const double data[28][28]){

    // printf("run forward\n");

    
    float input[28][28];

    for (int i = 0; i<28; i++){
        for (int j = 0; j<28; j++){
            input[i][j] = data[i][j];
        }
    }

    

    l_input.clear();
	l_c1.clear();
	l_s1.clear();
	l_f.clear();

    // printf("**************************************\n");


    //example for convLayer 1:

    l_input.setInput((float *)input);

    //printf("input image: %f\n", &l_input.output[0][0]);


    //timer
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    

    int bz;
    bz = ceil((float)28/TILE_WIDTH)*ceil((float)28/TILE_WIDTH);
    dim3 gridDim(1, 6, bz);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
    ConvLayerForward_Kernel_1<<<gridDim,blockDim>>>((float (*)[28])l_input.output, (float (*)[24][24])l_c1.preact, (float (*)[5][5])l_c1.weight, l_c1.bias, 1, 28, 28, 24, 5, 6);
    apply_sigmoid <<<64,64>>>(l_c1.preact, l_c1.output, l_c1.bytes);

    // for pooling layer example:
    bz = ceil((float)6/TILE_WIDTH)*ceil((float)6/TILE_WIDTH);
    dim3 gridDimPool(1, 6, bz);
    dim3 blockDimPool(TILE_WIDTH, TILE_WIDTH, 1);
    MaxPool2dForward_Kernel_1<<<gridDimPool,blockDimPool>>>((float (*)[24][24])l_c1.output, (float (*)[6][6])l_s1.preact, (float (*)[4][4])l_s1.weight, l_s1.bias ,24, 24, 6, 4);
    apply_sigmoid <<<64,64>>>(l_s1.preact, l_s1.output, l_s1.bytes);

    // for fully connected layer
    bz = ceil((float)10/TILE_WIDTH);
    dim3 gridDimfc(1, 10, 1);
    dim3 blockDimfc(6, 6, 6);
    FullyConLayerForward_kernel<<<gridDimfc,blockDimfc>>>((float (*)[6][6])l_s1.output, (float (*)[6][6][6])l_f.weight, l_f.preact, l_f.bias, 1, 6, 10, 1, 10);
	apply_sigmoid<<<64, 64>>>(l_f.preact, l_f.output, l_f.bytes);


    //end timer:
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop); // after hipEventRecord
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

    return time;
}

static float backward(){
    //timer
	hipEvent_t start2, stop2;
	float time;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
    
    dim3 gridDimfc(1, 10, 1);
    dim3 blockDimfc(6, 6, 6);
    bp_f<<<gridDimfc, blockDimfc>>>(
        (float (*)[6][6][6])l_f.d_weight, 
        l_f.d_preact,
        l_f.bias,
        (float (*)[6][6][6]) l_f.weight,
        (float (*)[6][6])l_s1.output,
        (float (*)[6][6])l_s1.d_output,
        (float (*)[6][6])l_s1.d_preact);

    
    dim3 gridDims(1, 6, 1);
    dim3 blockDims(6, 6, 1);
    bp_s1<<<gridDims, blockDims>>>(
        (float (*)[6][6])l_s1.preact,
        (float (*)[6][6])l_s1.d_output,
        (float (*)[6][6])l_s1.d_preact,
        (float (*)[4][4])l_s1.d_weight,
        (float (*)[4][4])l_s1.weight,
        (float (*)[24][24])l_c1.output,
        (float (*)[24][24])l_c1.d_output,
        l_s1.bias);

    
    dim3 gridDimc(1, 6, 1);
    dim3 blockDimc(24, 24, 1);
    bp_c1<<<gridDimc, blockDimc>>>(
        (float (*)[24][24])l_c1.preact,
        (float (*)[24][24])l_c1.d_preact,
        (float (*)[24][24])l_c1.d_output,
        (float (*)[5][5])l_c1.d_weight,
        (float (*)[5][5])l_c1.weight,
        (float (*)[28])l_input.output,
        l_c1.bias);

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2); // after hipEventRecord
    hipEventElapsedTime(&time, start2, stop2);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    return time;
}

static void learn(){

    float time_taken = 0.0;

    clock_t t;
	t = clock();

    for(int i=0; i< train_cnt; i++){
    //for(int i=0; i<10; i++){
    //     printf("label: %d \n", train_set[i].label);

        l_f.bp_clear();
		l_s1.bp_clear();
		l_c1.bp_clear();
        
        time_taken += forward(train_set[i].data);
        makeError<<<10, 1>>>(l_f.d_preact, l_f.output, train_set[i].label, 10);
        time_taken += backward();

     }

     printf("time on GPU: %.5f seconds\n", time_taken /  1000);

     t = clock() - t;
     float cpu_time = (float)t/CLOCKS_PER_SEC;
     printf("Total spend %.2f s.\n", cpu_time);
}


static unsigned int classify(double data[28][28])
{
	float res[10];

	forward(data);

	unsigned int max = 0;

    hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);
	// hipMemcpy(res, l_f.d_preact, sizeof(float) * 10, hipMemcpyDeviceToHost);

	for (int i = 1; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}

// Perform forward propagation of test data
static void test()
{
	int error = 0;

	for (int i = 0; i < test_cnt; ++i) {
		if (classify(test_set[i].data) != test_set[i].label) {
			++error;
		}
	}

	printf("Test Accuracy:: %.2lf%%\n", 100 - ( double(error) / double(test_cnt) * 100.0));
}


int main(){
    int epoch = 5;
    printf("CNN CUDA version result: \n");
    printf("Number of epoch: %d  \n\n", epoch);
    loadData();
    
    for (int i = 0; i < epoch; i++){
        printf("epoch: %d  \n", i + 1);
        learn();
        test();
    }
    
    
    printf("finish\n");

    return 0;
}